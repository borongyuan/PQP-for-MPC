#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>


#define NUM_ITER 1000

#define pHorizon 1
#define nState 29
#define nInput 7
#define nOutput 7
#define nDis 1

#define erc 1e-6
#define eac 1e-6
#define eaj 1e-6
#define erj 1e-6

__global__ void printMat(float *mat, int N, int M)
{
	printf("printing mat\n");
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<M;j++)
		{
			printf("%f ",mat[i*M+j]);
		}
		printf("\n");
	}
	printf("\n");
}



__global__ void initMatCuda(float *mat, float val, int N)								// parallel
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<N)	
	{
		mat[id] = val;
	}
}

void initMat(float *mat, float val, int N)								// parallel
{
	dim3 block = 1024;
	dim3 grid = (N+1024-1)/1024;

	initMatCuda<<<grid, block>>>(mat, val, N);
}



float *newMatrixCUDA(int n, int m)			
{
	float *tmp = NULL;
	
	hipError_t err = hipMalloc((void **)&tmp, n*m*sizeof(float));

	if ( err != hipSuccess )
	{
		printf (" Failed to allocate device matrix! %s\n", hipGetErrorString(err));
		exit ( EXIT_FAILURE ) ;
	}

	initMat(tmp, 0, n*m);
	return tmp;
}

float *newMatrix(int n, int m)			
{
	float *tmp = (float *)malloc(n*m*sizeof(float));
	for(int i=0;i<n*m;i++)
	{
		tmp[i] = 0;
	}
	return tmp;
}



void copyToDevice(float *dM, float *hM, int n, int m)
{
	int size = n*m;
	hipMemcpy (dM ,hM, size * sizeof ( float ) , hipMemcpyHostToDevice );
}

void copyToHost(float *hM, float *dM, int n, int m)
{
	int size = n*m;
	hipMemcpy (hM ,dM, size * sizeof ( float ) , hipMemcpyDeviceToHost );
}



__global__ void copyMatrixCuda(float *output, float *mat, int a, int b)		// parallel
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<a*b)	
	{
		output[id] = mat[id];
	}
}

void copyMatrix(float *output, float *mat, int a, int b)		// parallel
{
	dim3 block = 1024;
	dim3 grid = (a*b+1024-1)/1024;

	copyMatrixCuda<<<grid,block>>>(output, mat, a, b);
}



__global__ void transposeCuda(float *odata, float *idata, int n, int m)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(x<n && y<m)
		odata[y*n+x] = idata[x*m+y];
}

void transpose(float *odata, float *idata, int n, int m)
{
	dim3 block(32,32,1);
	dim3 grid((n+31)/32, (m+31)/32);
	
	transposeCuda<<<grid,block>>>(odata,idata,n,m);
}



__global__ void matrixMultiplyCuda(float *output, float *matrix1, float *matrix2, int a, int b, int c) 		//mat1-a*b	mat2-b*c
{		
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
		
	if(x<a && y<c)
	{
		float val = 0;
		for(int k=0;k<b;k++)
		{
			val += matrix1[x*b+k]*matrix2[k*c+y];
		}
		output[x*c+y] = val;
	}
}

void matrixMultiply(float *output, float *mat1, int transpose1, float *mat2, int transpose2, int a, int b, int c) 		//mat1-a*b	mat2-b*c 	// parallel
{
	float *tmp = newMatrixCUDA(a,c);
	
	float *matrix1;
	float *matrix2;
	
	if(transpose1)
	{
		matrix1 = newMatrixCUDA(a,b);
		transpose(matrix1, mat1, b,a);
	}
	else
	{
		matrix1 = mat1;
	}
	
	if(transpose2)
	{
		matrix2 = newMatrixCUDA(b,c);
		transpose(matrix2, mat1, c,b);
	}
	else
	{
		matrix2 = mat2;
	}
	
	dim3 block(32,32,1);
	dim3 grid((a+31)/32, (c+31)/32);
	matrixMultiplyCuda<<<grid, block>>>(output, matrix1, matrix2, a,b,c);
	
	if(transpose1)
	{
		hipFree(matrix1);
	}
	if(transpose2)
	{
		hipFree(matrix2);
	}

	hipFree(tmp);
}



__global__ void matrixAddCuda(float *A, float *B, float sign, int a, int b) 			// adds b to a 	// parallel
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<a*b)	
	{
		A[id] += sign * B[id];
	}
}

void matrixAdd(float *A, float *B, float sign, int a, int b) 			// adds b to a 	// parallel
{
	dim3 block = 1024;
	dim3 grid = (a*b+1024-1)/1024;

	matrixAddCuda<<<grid,block>>>(A,B,sign,a,b);
}



__global__ void negateMatrixCuda(float *mat, int n, int m)			// parallel
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<n*m)	
	{
		mat[id] = -mat[id];
	}
}

void negateMatrix(float *mat, int n, int m)			// parallel
{
	dim3 block = 1024;
	dim3 grid = (n*m+1024-1)/1024;

	negateMatrixCuda<<<grid,block>>>(mat,n,m);
}



__global__ void matrixPosCuda(float *mat1, float *mat2, int n, int m)			// parallel
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<n*m)	
	{
		mat1[id] = fmaxf(0.0, mat2[id]);

	}
}

void matrixPos(float *mat1, float *mat2, int n, int m)			// parallel
{
	dim3 block = 1024;
	dim3 grid = (n*m+1024-1)/1024;

	matrixPosCuda<<<grid,block>>>(mat1,mat2,n,m);
}



__global__ void matrixNegCuda(float *mat1, float *mat2, int n, int m)			// parallel
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<n*m)	
	{
		mat1[id] = fmaxf(0.0, -mat2[id]);
	}
}

void matrixNeg(float *mat1, float *mat2, int n, int m)			// parallel
{
	dim3 block = 1024;
	dim3 grid = (n*m+1024-1)/1024;

	matrixNegCuda<<<grid,block>>>(mat1,mat2,n,m);
}



__global__ void diagonalAddCuda(float *theta, float *tmp, int N)			// parallel
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<N)	
	{
		// printf("tmp %f\n",tmp[i]);
		theta[id*N+id] = fmaxf(tmp[id],5.0);
	}
}

void diagonalAdd(float *theta, float *tmp, int N)			// parallel
{
	dim3 block = 1024;
	dim3 grid = (N+1024-1)/1024;

	diagonalAddCuda<<<grid,block>>>(theta,tmp,N);
}



__global__ void compareCuda(float *GpU, float *Kp, int *re, int N)				// parallel
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<N)	
	{
		if(GpU[id] > Kp[id]+fmaxf(erc*Kp[id], eac))
		{
			*re = 0;
		}
	}
}

void compare(float *GpU, float *Kp, int *re, int N)				// parallel
{
	dim3 block = 1024;
	dim3 grid = (N+1024-1)/1024;

	compareCuda<<<grid,block>>>(GpU, Kp, re, N);
}




__global__ void updYCuda(float *Y_next, float *numerator, float *denominator, float *Y, int N)   // parallel
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	
	if(id<N)
	{
		Y_next[id] = numerator[id]/denominator[id]*Y[id];
	}
}

void updY(float *Y_next, float *numerator, float *denominator, float *Y, int N)   // parallel
{
	dim3 block = 1024;
	dim3 grid = (N+1023)/1024;
	
	updYCuda<<<grid, block>>>(Y_next, numerator, denominator, Y, N);
}




void Gauss_Jordan(float *A,float *res, int N)
{
    /*
    size=Size of input matrix
    A=input matrix
    res= inverted matrix
    */
    float temp;
    float *matrix = newMatrix(N, 2*N);

    for (int i = 0; i < N; i++) 
    { 
        for (int j = 0; j < 2 * N; j++) 
        { 
            matrix[i*2*N+j]=0;
            if (j == (i + N)) 
                matrix[i*2*N+j] = 1; 
        } 
    }

    for (int i = 0; i < N; i++) 
    { 
        for (int j = 0; j < N; j++) 
        { 
            matrix[i*2*N+j]=A[i*N+j];

        } 
    }

    for (int i = N - 1; i > 0; i--) 
    { 
        if (matrix[(i - 1)*2*N+0] < matrix[i*2*N+0]) 
            for (int j = 0; j < 2 * N; j++) 
            { 
                temp = matrix[i*2*N+j]; 
                matrix[i*2*N+j] = matrix[(i - 1)*2*N+j]; 
                matrix[(i - 1)*2*N+j] = temp; 
            } 
    }

    for (int i = 0; i < N; i++)
    { 

        for (int j = 0; j < N; j++) 
        { 
            if (j != i) 
            { 
                temp = matrix[j*2*N+i] / matrix[i*2*N+i]; 
                for (int k = 0; k < 2 * N; k++) 
                { 
                    matrix[j*2*N+k] -= matrix[i*2*N+k] * temp; 
                } 
            } 
        } 
    } 

    for (int i = 0; i < N; i++)
    { 
        temp = matrix[i*2*N+i]; 
        for (int j = 0; j < 2 * N; j++)
        {
            matrix[i*2*N+j] = matrix[i*2*N+j] / temp; 
        } 
    }

    for (int i = 0; i < N; i++) 
    { 
        for (int j = N; j <2*N; j++) 
        { 
            res[i*N+j-N]=matrix[i*2*N+j];

        } 
    }

    free(matrix);
}



void computeUfromY(float *U, float *Y, float *Fp, float *Gp, float *Qp_inv, int N, int M)
{
	float *tmp = newMatrixCUDA(M,1);
	matrixMultiply(tmp, Gp, 1, Y, 0, M, N, 1);
	matrixAdd(tmp, Fp, 1, M, 1);
	matrixMultiply(U, Qp_inv, 0, tmp, 0, M, M, 1);
	negateMatrix(U, M, 1);
	hipFree(tmp);
}

void computeFp(float *Fp, float *Fp1, float *Fp2, float *Fp3, float *D, float *x)
{
	matrixMultiply(Fp, Fp1, 0, D, 0, nInput*pHorizon, nDis*pHorizon, 1);
	float *Fp2x = newMatrixCUDA(nInput*pHorizon,1);
	matrixMultiply(Fp2x, Fp2, 0, x, 0, nInput*pHorizon, nState, 1);
	matrixAdd(Fp, Fp2x, 1, nInput*pHorizon, 1);
	matrixAdd(Fp, Fp3, -1, nInput*pHorizon, 1);
	
	hipFree(Fp2x);	
	// for(int i=0;i<nInput*pHorizon;i++)
	// {
	// 	printf("%f\n", Fp[i]);
	// }
	// printf("\n");
	// printf("%d\n", Fp);
}

void computeMp(float *Mp, float *Mp1, float *Mp2, float *Mp3, float *Mp4, float *Mp5, float *Mp6, float *D, float *x)
{
	initMat(Mp, 0, 1);

	float *tmp = newMatrixCUDA(1,nState);
	matrixMultiply(tmp, x, 1, Mp1, 0, 1, nState, nState);
	matrixMultiply(tmp, tmp, 0, x, 0, 1, nState, 1);

	matrixAdd(Mp, tmp, 0.5, 1,1);
//	printMat<<<1,1>>>(Mp, 1, 1);

	matrixMultiply(tmp, D, 1, Mp2, 0, 1, nDis*pHorizon, nState);
	matrixMultiply(tmp, tmp, 0, x, 0, 1, nState, 1);

	matrixAdd(Mp, tmp, 0.5, 1,1);
	
	matrixMultiply(tmp, Mp4, 1, x, 0, 1, nState, 1);

	matrixAdd(Mp, tmp, 0.5, 1,1);

	hipFree(tmp);
	tmp = newMatrixCUDA(1, nDis*pHorizon);
	matrixMultiply(tmp, D, 1, Mp3, 0, 1, nDis*pHorizon, nDis*pHorizon);
	matrixMultiply(tmp, tmp, 0, D, 0, 1, nDis*pHorizon, 1);

	matrixAdd(Mp, tmp, 0.5, 1,1);

	matrixMultiply(tmp, Mp5, 1, D, 0, 1, nDis*pHorizon, 1);

	matrixAdd(Mp, tmp, 0.5, 1,1);

	matrixAdd(Mp, Mp6, 0.5, 1,1);
	hipFree(tmp);
}

void computeQd(float *Qd, float *Gp_Qp_inv, float *Gp, int N, int M)
{
	matrixMultiply(Qd, Gp_Qp_inv, 0, Gp, 1, N, M, N);	
}

void computeFd(float *Fd, float *Gp_Qp_inv, float *Fp, float *Kp, int N, int M)
{
	matrixMultiply(Fd, Gp_Qp_inv, 0, Fp, 0, N, M, 1);
	matrixAdd(Fd, Kp, 1, N, 1);
}

void computeMd(float *Md, float *Fp, float* Qp_inv, float* Mp, int N, int M)
{
	float *tmp = newMatrixCUDA(1,M);
	matrixMultiply(tmp, Fp, 1, Qp_inv, 0, 1, M, M);
	matrixMultiply(Md, tmp, 0, Fp, 0, 1, M, 1);
	matrixAdd(Md, Mp, -1, 1, 1);
	hipFree(tmp);
}

void convertToDual(float *Qd, float *Fd, float *Md, float *Qp_inv, float *Gp, float *Kp, float *Fp, float *Mp, int N, int M)
{	
	float *Gp_Qp_inv = newMatrixCUDA(N,M);
	matrixMultiply(Gp_Qp_inv, Gp, 0, Qp_inv, 0, N, M, M);
	computeQd(Qd, Gp_Qp_inv, Gp, N, M);
	computeFd(Fd, Gp_Qp_inv, Fp, Kp, N, M);
	computeMd(Md, Fp, Qp_inv, Mp, N, M);

	hipFree(Gp_Qp_inv);
}

void computeTheta(float *theta, float *Qd, int N)
{
	float *Qdn = newMatrixCUDA(N,N);
	matrixNeg(Qdn, Qd, N, N);

	float *one = newMatrixCUDA(N,1);
	initMat(one, 1, N);

	float *tmp = newMatrixCUDA(N,1);
	matrixMultiply(tmp, Qdn, 0, one, 0, N,N,1);

	diagonalAdd(theta, tmp, N);

	hipFree(Qdn);
	hipFree(one);
	hipFree(tmp);
}

void computeQdp_theta(float *Qdp_theta, float *Qd, float *theta, int N)
{
	matrixPos(Qdp_theta, Qd, N, N);
	matrixAdd(Qdp_theta, theta, 1, N, N);
}

void computeQdn_theta(float *Qdn_theta, float *Qd, float *theta, int N)
{
	matrixNeg(Qdn_theta, Qd, N, N);
	matrixAdd(Qdn_theta, theta, 1, N, N);
}

void computealphaY(float *alphaY, float *ph, float *Qd, float *Y, float *Fd, int N)
{
	float *temp = newMatrixCUDA(1,N);

	matrixMultiply(temp, ph, 1, Qd, 0, 1, N, N);
	matrixMultiply(temp, temp, 0, ph, 0, 1, N, 1);

	if(temp[0] > 0)
	{
		float *temp2 = newMatrixCUDA(1,N);

		matrixMultiply(temp2, Y, 1, Qd, 0, 1, N, N);
		
		matrixAdd(temp2, Fd, 1, 1, N);
		
		matrixMultiply(temp2, temp2, 0, ph, 0, 1, N, 1);

		*alphaY = -temp2[0]/temp[0];

		hipFree(temp2);
	}
	else
	{
		alphaY = 0;
	}

	hipFree(temp);
}

void updateY1(float *Y_next, float *Y, float alphaY, float *ph, int N)
{
	copyMatrix(Y_next, Y, N, 1);
	matrixAdd(Y_next, ph, alphaY, N, 1);
}

void updateY2(float *Y_next, float *Y, float *Qdp_theta, float *Qdn_theta, float *Fd, float *Fdp, float *Fdn, int N)
{
	float *numerator = newMatrixCUDA(N,1);
	float *denominator = newMatrixCUDA(N,1);

	matrixMultiply(numerator, Qdn_theta, 0, Y, 0, N, N, 1);
	matrixMultiply(denominator, Qdp_theta, 0, Y, 0, N, N, 1);

	matrixAdd(numerator, Fdn, 1, N, 1);
	matrixAdd(denominator, Fdp, 1, N, 1);

	updY(Y_next, numerator, denominator, Y, N);

	hipFree(numerator);
	hipFree(denominator);
}

void computeph(float *ph, float *Qd, float *Y, float *Fd, int N)
{
	matrixMultiply(ph, Qd, 0, Y, 0, N, N, 1);
	matrixAdd(ph, ph, 1, N, 1);
	matrixNeg(ph, ph, N, 1);
}

int checkFeas(float *U, float *Gp, float *Kp, int N, int M)
{
	float *tmp = newMatrixCUDA(N,1);
	matrixMultiply(tmp, Gp, 0, U, 0, N, M, 1);
	int re = 1;
	compare(tmp, Kp, &re, N);

	hipFree(tmp);
	return re;
}	

float computeCost(float *Z, float *Q, float *F, float *M, int N)
{
	float *J=newMatrixCUDA(1,1);

	float *tmp = newMatrixCUDA(1,N);
	matrixMultiply(tmp, Z, 1, Q, 0, 1, N, N);
	matrixMultiply(tmp, tmp, 0, Z, 0, 1, N, 1);

	matrixAdd(J, tmp, 0.5, 1,1);

	matrixMultiply(tmp, F, 1, Z, 0, 1, N, 1);

	matrixAdd(J, tmp, 1, 1,1);
//	printMat<<<1,1>>>(J,1,1);
//	printMat<<<1,1>>>(M,1,1);
	matrixAdd(J, M, 0.5, 1,1);
	

	float *hJ = newMatrix(1,1);
	copyToHost(hJ,J,1,1);

	float cost = hJ[0];
	free(hJ);
	hipFree(J);
	hipFree(tmp);

	return cost;
}

int terminate(float *Y, float *Qd, float *Fd, float *Md, float *U, float *Qp, float *Qp_inv, float *Fp, float *Mp, float *Gp, float *Kp, int N, int M)
{
	computeUfromY(U, Y, Fp, Gp, Qp_inv, N, M);

	if(!checkFeas(U, Gp, Kp, N, M))	return 0;

	float Jd = computeCost(Y, Qd, Fd, Md, N);
	float Jp = computeCost(U, Qp, Fp, Mp, M);

	if(Jp>-Jd)	return 0;
	if(Jp+Jd>eaj)	return 0;
	if((Jp+Jd)/fabs(Jd)>erj) return 0;

	return 1;
}

void solveQuadraticDual(float *Y, float *Qd, float *Fd, float *Md, float *U, float *Qp, float *Qp_inv, float *Fp, float *Mp, float *Gp, float *Kp, int N, int M)
{
	float *theta = newMatrixCUDA(N,N);
	float *Qdp_theta = newMatrixCUDA(N,N);
	float *Qdn_theta = newMatrixCUDA(N,N);
	float *Y_next = newMatrixCUDA(N,1);
	
	float *Fdn = newMatrixCUDA(N,1);
	float *Fdp = newMatrixCUDA(N,1);

	matrixPos(Fdp, Fd, N, 1);
	matrixNeg(Fdn, Fd, N, 1);
	
	computeTheta(theta, Qd, N);
	computeQdp_theta(Qdp_theta, Qd, theta, N);
	computeQdn_theta(Qdn_theta, Qd, theta, N);

	initMat(Y, 1000.0, N);
	// for(int i=0;i<N;i++) Y[i] = i+1;

	float *ph = newMatrixCUDA(N,1);
	long int h=1;
	float alphaY=0;

//	 while(h<NUM_ITER)
	while(!terminate(Y, Qd, Fd, Md, U, Qp, Qp_inv, Fp, Mp, Gp, Kp, N, M))
	{	
		// if(h>100000) break;
//		 printf("h %ld\n",h);
		if(1)
		{
			//update
			// printf("here\n");
			updateY2(Y_next, Y, Qdp_theta, Qdn_theta, Fd, Fdp, Fdn, N);			
			// printf("there\n");
		}
		else
		{
			// printf("accelerating\n");
			// accelerate
			computeph(ph, Qd, Y, Fd, N);
			computealphaY(&alphaY, ph, Qd, Y, Fd, N);
			// printf("alpha %f\n", alphaY);
			
			updateY1(Y_next, Y, alphaY/10, ph, N);

		}

		copyMatrix(Y, Y_next, N, 1);
		// for(int i=0;i<N;i++)
		// {
		// 	printf("%f ",Y[i]);
		// }
		// printf("\n\n");

		h++;
	}
	printf("Printing number of iterations = %ld\n",h);

	hipFree(theta);
	hipFree(Qdp_theta);
	hipFree(Qdn_theta);
	hipFree(Y_next);
	hipFree(ph);
	hipFree(Fdp);
	hipFree(Fdn);
}

void input(float* qp_inv, float* Fp1, float* Fp2, float * Fp3, float * Mp1, float * Mp2, float * Mp3, float* Mp4, float* Mp5, float* Mp6, float* Gp, float* Kp, float* x, float* D, float* theta, float* Z)
{
	FILE *fptr;
	int i,j;
	float num;

	//Fill Qp_inverse	
	fptr = fopen("./example/Qp_inv.txt","r");
	for(i=0;i<pHorizon*nInput;i++)
	{
		for(j=0;j<pHorizon*nInput;j++)
		{
			fscanf(fptr,"%f", &num);
			qp_inv[j*pHorizon*nInput+i] = num;
		}
	}
	fclose(fptr);

	//Fill Fp1
	fptr = fopen("./example/Fp1.txt","r");
	for(i=0;i<nDis*pHorizon;i++)
	{
		for(j=0;j<nInput*pHorizon;j++)
		{
			fscanf(fptr,"%f", &num);
			Fp1[j*nDis*pHorizon+i] = num;
		}
	}
	fclose(fptr);

	//Fill Fp2
	fptr = fopen("./example/Fp2.txt","r");
	for(i=0;i<nState;i++)
	{
		for(j=0;j<nInput*pHorizon;j++)
		{
			fscanf(fptr,"%f", &num);
			Fp2[j*nState+i] = num;
		}
	}
	fclose(fptr);

	//Fill Fp3
	fptr = fopen("./example/Fp3.txt","r");
	for(j=0;j<nInput*pHorizon;j++)
	{
		fscanf(fptr,"%f", &num);
		Fp3[j] = num;
	}
	fclose(fptr);

	//Fill Mp1
	fptr = fopen("./example/Mp1.txt","r");
	for(i=0;i<nState;i++)
	{
		for(j=0;j<nState;j++)
		{
			fscanf(fptr,"%f", &num);
			Mp1[j*nState+i] = num;
		}
	}
	fclose(fptr);

	//Fill Mp2
	fptr = fopen("./example/Mp2.txt","r");
	for(i=0;i<nState;i++)
	{
		for(j=0;j<nDis*pHorizon;j++)
		{
			fscanf(fptr,"%f", &num);
			Mp2[j*nState+i] = num;
		}
	}
	fclose(fptr);

	//Fill Mp3
	fptr = fopen("./example/Mp3.txt","r");
	for(i=0;i<nDis*pHorizon;i++)
	{
		for(j=0;j<nDis*pHorizon;j++)
		{
			fscanf(fptr,"%f", &num);
			Mp3[j*nDis*pHorizon+i] = num;
		}
	}
	fclose(fptr);

	//Fill Mp4
	fptr = fopen("./example/Mp4.txt","r");
	for(i=0;i<nState;i++)
	{
		fscanf(fptr,"%f", &num);
		Mp4[i] = num;
	}
	fclose(fptr);

	//Fill Mp5
	fptr = fopen("./example/Mp5.txt","r");
	for(i=0;i<nDis*pHorizon;i++)
	{
		fscanf(fptr,"%f", &num);
		Mp5[i] = num;
	}
	fclose(fptr);

	//Fill Mp6
	fptr = fopen("./example/Mp6.txt","r");
	fscanf(fptr,"%f", &num);
	Mp6[0] = num;
	fclose(fptr);

	//Fill Gp
	fptr = fopen("./example/Gp.txt","r");
	for(i=0;i<pHorizon*nInput;i++)
	{
		for(j=0;j<4*pHorizon*nInput;j++)
		{
			fscanf(fptr,"%f", &num);
			Gp[j*pHorizon*nInput+i] = num;
		}
	}
	fclose(fptr);

	//Fill Kp
	fptr = fopen("./example/Kp.txt","r");
	for(i=0;i<4*pHorizon*nInput;i++)
	{
		fscanf(fptr,"%f", &num);
		Kp[i] = num;
	}
	fclose(fptr);

	//Fill Z
	fptr = fopen("./example/Z.txt","r");
	for(i=0;i<nState;i++)
	{
		for(j=0;j<nOutput*pHorizon;j++)
		{
			fscanf(fptr,"%f", &num);
			Z[j*nState+i] = num;
		}
	}
	fclose(fptr);

	//Fill Theta
	fptr = fopen("./example/Theta.txt","r");
	for(i=0;i<nDis*pHorizon;i++)
	{
		for(j=0;j<nOutput*pHorizon;j++)
		{
			fscanf(fptr,"%f", &num);
			theta[j*nDis*pHorizon+i] = num;
		}
	}
	fclose(fptr);

	//Fill D
	fptr = fopen("./example/D.txt","r");
	for(i=0;i<nDis*pHorizon;i++)
	{
		fscanf(fptr,"%f", &num);
		D[i] = num;
	}
	fclose(fptr);

	//Fill x
	fptr = fopen("./example/x.txt","r");
	for(i=0;i<nState;i++)
	{
		fscanf(fptr,"%f", &num);
		x[i] = num;
	}
	fclose(fptr);
}

int main()
{
	// QP is of parametric from 
	// J(U) = min U 1/2*U'QpU + Fp'U + 1/2*Mp
	// st GpU <= Kp
	
	hipDeviceReset();
	 
	int N, M;

	M = pHorizon*nInput;
	N = 4*pHorizon*nInput;

	// host matrix
	float *hQp_inv = newMatrix(M,M);
	float *hQp = newMatrix(M,M);

	float *hFp1;
	float *hFp2;
	float *hFp3;

	float *hMp1;
	float *hMp2;
	float *hMp3;
	float *hMp4;
	float *hMp5;
	float *hMp6;

	float *hFp = newMatrix(nInput*pHorizon,1);
	float *hMp = newMatrix(1,1);
	float *hGp;
	float *hKp;
	float *hx;
	float *hD; 
	float *htheta; 
	float *hZ; 

	hFp1 = newMatrix(nInput*pHorizon, nDis*pHorizon);
	hFp2 = newMatrix(nInput*pHorizon, nState);
	hFp3 = newMatrix(1, nInput*pHorizon);
	hMp1 = newMatrix(nState, nState);
	hMp2 = newMatrix(nDis*pHorizon, nState);
	hMp3 = newMatrix(nDis*pHorizon, nDis*pHorizon);
	hMp4 = newMatrix(1, nState);
	hMp5 = newMatrix(1, nDis*pHorizon);
	hMp6 = newMatrix(1,1);
	hGp = newMatrix(4*pHorizon*nInput, nInput*pHorizon);
	hKp = newMatrix(1,4*pHorizon*nInput);
	hZ = newMatrix(nOutput*pHorizon, nState);
	htheta = newMatrix(nOutput*pHorizon, nDis*pHorizon);
	hD = newMatrix(nDis*pHorizon,1);
	hx = newMatrix(nState, 1);

	// device matrix
	float *Qp_inv = newMatrixCUDA(M,M);
	float *Qp = newMatrixCUDA(M,M);

	float *Fp1;
	float *Fp2;
	float *Fp3;

	float *Mp1;
	float *Mp2;
	float *Mp3;
	float *Mp4;
	float *Mp5;
	float *Mp6;

	float *Fp = newMatrixCUDA(nInput*pHorizon,1);
	float *Mp = newMatrixCUDA(1,1);
	float *Gp;
	float *Kp;
	float *x;
	float *D; 
	float *theta; 
	float *Z; 

	Fp1 = newMatrixCUDA(nInput*pHorizon, nDis*pHorizon);
	Fp2 = newMatrixCUDA(nInput*pHorizon, nState);
	Fp3 = newMatrixCUDA(1, nInput*pHorizon);
	Mp1 = newMatrixCUDA(nState, nState);
	Mp2 = newMatrixCUDA(nDis*pHorizon, nState);
	Mp3 = newMatrixCUDA(nDis*pHorizon, nDis*pHorizon);
	Mp4 = newMatrixCUDA(1, nState);
	Mp5 = newMatrixCUDA(1, nDis*pHorizon);
	Mp6 = newMatrixCUDA(1,1);
	Gp = newMatrixCUDA(4*pHorizon*nInput, nInput*pHorizon);
	Kp = newMatrixCUDA(1,4*pHorizon*nInput);
	Z = newMatrixCUDA(nOutput*pHorizon, nState);
	theta = newMatrixCUDA(nOutput*pHorizon, nDis*pHorizon);
	D = newMatrixCUDA(nDis*pHorizon,1);
	x = newMatrixCUDA(nState, 1);	

	input(hQp_inv, hFp1, hFp2, hFp3, hMp1, hMp2, hMp3, hMp4, hMp5, hMp6, hGp, hKp, hx, hD, htheta, hZ);
	Gauss_Jordan(hQp_inv, hQp, M);
	copyToDevice(Qp_inv, hQp_inv, M, M);
	copyToDevice(Qp, hQp, M, M);
	copyToDevice(Fp1, hFp1, nInput*pHorizon, nDis*pHorizon);
	copyToDevice(Fp2, hFp2, nInput*pHorizon, nState);
	copyToDevice(Fp3, hFp3, 1, nInput*pHorizon);
	copyToDevice(Mp1, hMp1, nState, nState);
	copyToDevice(Mp2, hMp2, nDis*pHorizon, nState);
	copyToDevice(Mp3, hMp3, nDis*pHorizon, nDis*pHorizon);
	copyToDevice(Mp4, hMp4, 1, nState);
	copyToDevice(Mp5, hMp5, 1, nDis*pHorizon);
	copyToDevice(Mp6, hMp6, 1,1);
	copyToDevice(Gp, hGp, 4*pHorizon*nInput, nInput*pHorizon);
	copyToDevice(Kp, hKp, 1,4*pHorizon*nInput);
	copyToDevice(Z, hZ, nOutput*pHorizon, nState);
	copyToDevice(D, hD, nDis*pHorizon,1);
	copyToDevice(theta, htheta, nOutput*pHorizon, nDis*pHorizon);
	copyToDevice(x, hx, nState, 1);

	computeFp(Fp, Fp1, Fp2, Fp3, D, x);
	computeMp(Mp, Mp1, Mp2, Mp3, Mp4, Mp5, Mp6, D, x);
	// printf("Mp %f\n", Mp[0]);
	// printf("er\n");
//	printMat<<<1,1>>>(Mp,1,1);
	// matrices and vectors required for dual form of QP
	float *Qd = newMatrixCUDA(N,N);
	float *Fd = newMatrixCUDA(N,1);
	float *Md = newMatrixCUDA(1,1);	
	float *Y  = newMatrixCUDA(N,1);
	float *U  = newMatrixCUDA(M,1);
	// printf("er\n");
	convertToDual(Qd, Fd, Md, Qp_inv, Gp, Kp, Fp, Mp, N, M);
	// printf("Qd\n");
	// for(int i=0;i<N;i++)
	// {
	// 	for(int j=0;j<N;j++)
	// 	{
	// 		printf("%f ", Qd[i*N+j]);
	// 	}
	// 	printf("\n");
	// }
	// printf("Fd\n");
	// printf("%f\n", Md[0]);
	// for(int i=0;i<N;i++)
	// {
	// 	printf("%f ", Fp[i]);
	// }
	// printf("\n");
	solveQuadraticDual(Y, Qd, Fd, Md, U, Qp, Qp_inv, Fp, Mp, Gp, Kp, N, M);
	// printf("erer\n");

	computeUfromY(U, Y, Fp, Gp, Qp_inv, N, M);

	// U[0] = -6.399018;
	// U[1] = -10.648726;
	// U[2] = -4.792378;
	// U[3] = -7.033428;
	// U[4] = -4.792378;
	// U[5] = -10.648726;
	// U[6] = -6.399018;

	// U[0] = -6.398985;
	// U[1] = -10.646729;
	// U[2] = -4.792132;
	// U[3] = -7.027614;
	// U[4] = -4.792255;
	// U[5] = -10.643004;
	// U[6] = -6.398996;

	float Jp = computeCost(U, Qp, Fp, Mp, M);
	float Jd = computeCost(Y, Qd, Fd, Md, N);

	printf("Jp = %f\n", Jp);
	printf("Jd = %f\n", Jd);
	
	float *hU = newMatrix(M,1);
	float *hY = newMatrix(N,1);

	copyToHost(hU,U,M,1);
	copyToHost(hY,Y,N,1);

	// printf("Printing Y*\n");
	// for(int i=0;i<N;i++)
	// {
	// 	printf("%f\n", hY[i]);
	// }
	printf("Printing U*\n");
	for(int i=0;i<M;i++)
	{
		printf("\t%f\n", hU[i]);
	}

	free(hQp_inv);
	free(hQp);
	free(hFp1);
	free(hFp2);
	free(hFp3);
	free(hMp1);
	free(hMp2);
	free(hMp3);
	free(hMp4);
	free(hMp5);
	free(hMp6); 
	free(hFp);
	free(hMp);
	free(hGp);
	free(hKp);
	free(hx);
	free(hD);
	free(htheta);
	free(hZ);

	hipFree(Qp_inv);
	hipFree(Qp);
	hipFree(Fp1);
	hipFree(Fp2);
	hipFree(Fp3);
	hipFree(Mp1);
	hipFree(Mp2);
	hipFree(Mp3);
	hipFree(Mp4);
	hipFree(Mp5);
	hipFree(Mp6); 
	hipFree(Fp);
	hipFree(Mp);
	hipFree(Gp);
	hipFree(Kp);
	hipFree(x);
	hipFree(D);
	hipFree(theta);
	hipFree(Z);
	
	hipFree(Qd);
	hipFree(Fd);
	hipFree(Md);
	hipFree(Y);
	hipFree(U);
}
