/**************************************************************************
* This file contains implementation of pqp (parallel quadratic programming)
* GPU version unoptimised (Basic) for MPC Term Project of HP3 Course.
* Group 7 CSE Dept. IIT KGP
*	Objective function: 1/2 U'QpU + Fp'U + 1/2 Mp
*	Constraints: GpU <= Kp
**************************************************************************/
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>


#define NUM_ITER 100

#define pHorizon 1
#define nState 29
#define nInput 7
#define nOutput 7
#define nDis 1

#define erc 7
#define eac 100000
#define eaj 100000
#define erj 7

__global__ void printMat(float *mat, int N, int M)
{
	printf("printing mat\n");
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<M;j++)
		{
			printf("%f ",mat[i*M+j]);
		}
		printf("\n");
	}
	printf("\n");
}



__global__ void initMatCuda(float *mat, float val, int N)								
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<N)	
	{
		mat[id] = val;
	}
}

/**************************************************************************
* This is utility function initialize the matrix
*   1. Parameter is float type matrix pointer (*mat), float val, 
*		size of matrix 
*   2. Return type void
**************************************************************************/
void initMat(float *mat, float val, int N)								
{
	dim3 block = 1024;
	dim3 grid = (N+1024-1)/1024;

	initMatCuda<<<grid, block>>>(mat, val, N);
}



float *newMatrixCUDA(int n, int m)			
{
	float *tmp = NULL;
	
	hipError_t err = hipMalloc((void **)&tmp, n*m*sizeof(float));

	if ( err != hipSuccess )
	{
		printf (" Failed to allocate device matrix! %s\n", hipGetErrorString(err));
		exit ( EXIT_FAILURE ) ;
	}

	initMat(tmp, 0, n*m);
	return tmp;
}
/**************************************************************************
* This is utility function for create new  matrix
*   1. Parameter is (int n, int m) dimension of (n X m matrix) , 
*	2. Return pointer of new matrix
*   3. This function create dynamic size matrix using malloc
**************************************************************************/
float *newMatrix(int n, int m)			
{
	float *tmp = (float *)malloc(n*m*sizeof(float));
	for(int i=0;i<n*m;i++)
	{
		tmp[i] = 0;
	}
	return tmp;
}



void copyToDevice(float *dM, float *hM, int n, int m)
{
	int size = n*m;
	hipMemcpy (dM ,hM, size * sizeof ( float ) , hipMemcpyHostToDevice );
}

void copyToHost(float *hM, float *dM, int n, int m)
{
	int size = n*m;
	hipMemcpy (hM ,dM, size * sizeof ( float ) , hipMemcpyDeviceToHost );
}



__global__ void copyMatrixCuda(float *output, float *mat, int a, int b)		
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<a*b)	
	{
		output[id] = mat[id];
	}
}

void copyMatrix(float *output, float *mat, int a, int b)		
{
	dim3 block = 1024;
	dim3 grid = (a*b+1024-1)/1024;

	copyMatrixCuda<<<grid,block>>>(output, mat, a, b);
}



__global__ void transposeCuda(float *odata, float *idata, int n, int m)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(x<n && y<m)
		odata[y*n+x] = idata[x*m+y];
}

void transpose(float *odata, float *idata, int n, int m)
{
	dim3 block(32,32,1);
	dim3 grid((n+31)/32, (m+31)/32);
	
	transposeCuda<<<grid,block>>>(odata,idata,n,m);
}



__global__ void matrixMultiplyCuda(float *output, float *matrix1, float *matrix2, int a, int b, int c) 		//mat1-a*b	mat2-b*c
{		
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
		
	if(x<a && y<c)
	{
		float val = 0;
		for(int k=0;k<b;k++)
		{
			val += matrix1[x*b+k]*matrix2[k*c+y];
		}
		output[x*c+y] = val;
	}
}

void matrixMultiply(float *output, float *mat1, int transpose1, float *mat2, int transpose2, int a, int b, int c) 		//mat1-a*b	mat2-b*c 	
{
	float *tmp = newMatrixCUDA(a,c);
	
	float *matrix1;
	float *matrix2;
	
	if(transpose1 && a!=1 && b!=1)
	{
		matrix1 = newMatrixCUDA(a,b);
		transpose(matrix1, mat1, b,a);
			
	}
	else
	{
		matrix1 = mat1;
	}
	
	if(transpose2 && b!=1 && c!=1)
	{
		matrix2 = newMatrixCUDA(b,c);
		transpose(matrix2, mat1, c,b);
		
	}
	else
	{
		matrix2 = mat2;
	}
	
	dim3 block(32,32,1);
	dim3 grid((a+31)/32, (c+31)/32);
	matrixMultiplyCuda<<<grid, block>>>(output, matrix1, matrix2, a, b, c);
	
	if(transpose1 && a!=1 && b!=1)
	{
		hipFree(matrix1);
	}
	if(transpose2 && b!=1 && c!=1)
	{
		hipFree(matrix2);
	}

	hipFree(tmp);
}

/**************************************************************************
* This is utility function for generating addition or substraction 
*	of two matrix
*   1. Parameter is (pointer of matrix1, pointer of matrix2, float sign,int n int m)
*		dimension of (n X m matrix) 
*	2. sign parameters for decide addition or substraction
*	3. Result write back in matrix1
**************************************************************************/

__global__ void matrixAddCuda(float *A, float *B, float sign, int a, int b) 			// adds b to a 	
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<a*b)	
	{
		A[id] += sign * B[id];
	}
}

void matrixAdd(float *A, float *B, float sign, int a, int b) 			// adds b to a 	
{
	dim3 block = 1024;
	dim3 grid = (a*b+1024-1)/1024;

	matrixAddCuda<<<grid,block>>>(A,B,sign,a,b);
}



__global__ void negateMatrixCuda(float *mat, int n, int m)			
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<n*m)	
	{
		mat[id] = -mat[id];
	}
}

void negateMatrix(float *mat, int n, int m)			
{
	dim3 block = 1024;
	dim3 grid = (n*m+1024-1)/1024;

	negateMatrixCuda<<<grid,block>>>(mat,n,m);
}



__global__ void matrixPosCuda(float *mat1, float *mat2, int n, int m)			
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<n*m)	
	{
		mat1[id] = fmaxf(0.0, mat2[id]);

	}
}

void matrixPos(float *mat1, float *mat2, int n, int m)			
{
	dim3 block = 1024;
	dim3 grid = (n*m+1024-1)/1024;

	matrixPosCuda<<<grid,block>>>(mat1,mat2,n,m);
}



__global__ void matrixNegCuda(float *mat1, float *mat2, int n, int m)			
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<n*m)	
	{
		mat1[id] = fmaxf(0.0, -mat2[id]);
	}
}

void matrixNeg(float *mat1, float *mat2, int n, int m)			
{
	dim3 block = 1024;
	dim3 grid = (n*m+1024-1)/1024;

	matrixNegCuda<<<grid,block>>>(mat1,mat2,n,m);
}



__global__ void diagonalAddCuda(float *theta, float *tmp, int N)			
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<N)	
	{
		theta[id*N+id] = fmaxf(tmp[id],5.0);
	}
}

void diagonalAdd(float *theta, float *tmp, int N)			
{
	dim3 block = 1024;
	dim3 grid = (N+1024-1)/1024;

	diagonalAddCuda<<<grid,block>>>(theta,tmp,N);
}



__global__ void compareCuda(float *GpU, float *Kp, int *re, int N)				
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<N)	
	{
		if(GpU[id] > Kp[id]+fmaxf(erc*Kp[id], eac))
		{
			*re = 0;
		}
	}
}

void compare(float *GpU, float *Kp, int *re, int N)				
{
	dim3 block = 1024;
	dim3 grid = (N+1024-1)/1024;

	compareCuda<<<grid,block>>>(GpU, Kp, re, N);
}




__global__ void updYCuda(float *Y_next, float *numerator, float *denominator, float *Y, int N)   
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	
	if(id<N)
	{
		Y_next[id] = numerator[id]/denominator[id]*Y[id];
	}
}

void updY(float *Y_next, float *numerator, float *denominator, float *Y, int N)   
{
	dim3 block = 1024;
	dim3 grid = (N+1023)/1024;
	
	updYCuda<<<grid, block>>>(Y_next, numerator, denominator, Y, N);
}




void Gauss_Jordan(float *A,float *res, int N)
{
    /*
    size=Size of input matrix
    A=input matrix
    res= inverted matrix
    */
    float temp;
    float *matrix = newMatrix(N, 2*N);

    for (int i = 0; i < N; i++) 
    { 
        for (int j = 0; j < 2 * N; j++) 
        { 
            matrix[i*2*N+j]=0;
            if (j == (i + N)) 
                matrix[i*2*N+j] = 1; 
        } 
    }

    for (int i = 0; i < N; i++) 
    { 
        for (int j = 0; j < N; j++) 
        { 
            matrix[i*2*N+j]=A[i*N+j];

        } 
    }

    for (int i = N - 1; i > 0; i--) 
    { 
        if (matrix[(i - 1)*2*N+0] < matrix[i*2*N+0]) 
            for (int j = 0; j < 2 * N; j++) 
            { 
                temp = matrix[i*2*N+j]; 
                matrix[i*2*N+j] = matrix[(i - 1)*2*N+j]; 
                matrix[(i - 1)*2*N+j] = temp; 
            } 
    }

    for (int i = 0; i < N; i++)
    { 

        for (int j = 0; j < N; j++) 
        { 
            if (j != i) 
            { 
                temp = matrix[j*2*N+i] / matrix[i*2*N+i]; 
                for (int k = 0; k < 2 * N; k++) 
                { 
                    matrix[j*2*N+k] -= matrix[i*2*N+k] * temp; 
                } 
            } 
        } 
    } 

    for (int i = 0; i < N; i++)
    { 
        temp = matrix[i*2*N+i]; 
        for (int j = 0; j < 2 * N; j++)
        {
            matrix[i*2*N+j] = matrix[i*2*N+j] / temp; 
        } 
    }

    for (int i = 0; i < N; i++) 
    { 
        for (int j = N; j <2*N; j++) 
        { 
            res[i*N+j-N]=matrix[i*2*N+j];

        } 
    }

    free(matrix);
}



void computeUfromY(float *U, float *Y, float *Fp, float *Gp, float *Qp_inv, int N, int M)
{
	float *tmp = newMatrixCUDA(M,1);
	matrixMultiply(tmp, Gp, 1, Y, 0, M, N, 1);
	matrixAdd(tmp, Fp, 1, M, 1);
	matrixMultiply(U, Qp_inv, 0, tmp, 0, M, M, 1);
	negateMatrix(U, M, 1);
	hipFree(tmp);
}

void computeFp(float *Fp, float *Fp1, float *Fp2, float *Fp3, float *D, float *x)
{
	matrixMultiply(Fp, Fp1, 0, D, 0, nInput*pHorizon, nDis*pHorizon, 1);
	float *Fp2x = newMatrixCUDA(nInput*pHorizon,1);
	matrixMultiply(Fp2x, Fp2, 0, x, 0, nInput*pHorizon, nState, 1);
	matrixAdd(Fp, Fp2x, 1, nInput*pHorizon, 1);
	matrixAdd(Fp, Fp3, -1, nInput*pHorizon, 1);
	
	hipFree(Fp2x);	
}

void computeMp(float *Mp, float *Mp1, float *Mp2, float *Mp3, float *Mp4, float *Mp5, float *Mp6, float *D, float *x)
{
	initMat(Mp, 0, 1);

	float *tmp = newMatrixCUDA(1,nState);
	matrixMultiply(tmp, x, 1, Mp1, 0, 1, nState, nState);
	matrixMultiply(tmp, tmp, 0, x, 0, 1, nState, 1);

	matrixAdd(Mp, tmp, 0.5, 1,1);
//	printMat<<<1,1>>>(Mp, 1, 1);

	matrixMultiply(tmp, D, 1, Mp2, 0, 1, nDis*pHorizon, nState);
	matrixMultiply(tmp, tmp, 0, x, 0, 1, nState, 1);

	matrixAdd(Mp, tmp, 0.5, 1,1);
	
	matrixMultiply(tmp, Mp4, 1, x, 0, 1, nState, 1);

	matrixAdd(Mp, tmp, 0.5, 1,1);

	hipFree(tmp);
	tmp = newMatrixCUDA(1, nDis*pHorizon);
	matrixMultiply(tmp, D, 1, Mp3, 0, 1, nDis*pHorizon, nDis*pHorizon);
	matrixMultiply(tmp, tmp, 0, D, 0, 1, nDis*pHorizon, 1);

	matrixAdd(Mp, tmp, 0.5, 1,1);

	matrixMultiply(tmp, Mp5, 1, D, 0, 1, nDis*pHorizon, 1);

	matrixAdd(Mp, tmp, 0.5, 1,1);

	matrixAdd(Mp, Mp6, 0.5, 1,1);
	hipFree(tmp);
}

void computeQd(float *Qd, float *Gp_Qp_inv, float *Gp, int N, int M)
{
	matrixMultiply(Qd, Gp_Qp_inv, 0, Gp, 1, N, M, N);	
}

void computeFd(float *Fd, float *Gp_Qp_inv, float *Fp, float *Kp, int N, int M)
{
	matrixMultiply(Fd, Gp_Qp_inv, 0, Fp, 0, N, M, 1);
	matrixAdd(Fd, Kp, 1, N, 1);
}

void computeMd(float *Md, float *Fp, float* Qp_inv, float* Mp, int N, int M)
{
	float *tmp = newMatrixCUDA(1,M);
	matrixMultiply(tmp, Fp, 1, Qp_inv, 0, 1, M, M);
	matrixMultiply(Md, tmp, 0, Fp, 0, 1, M, 1);
	matrixAdd(Md, Mp, -1, 1, 1);
	hipFree(tmp);
}

void convertToDual(float *Qd, float *Fd, float *Md, float *Qp_inv, float *Gp, float *Kp, float *Fp, float *Mp, int N, int M)
{	
	float *Gp_Qp_inv = newMatrixCUDA(N,M);
	matrixMultiply(Gp_Qp_inv, Gp, 0, Qp_inv, 0, N, M, M);
	computeQd(Qd, Gp_Qp_inv, Gp, N, M);
	computeFd(Fd, Gp_Qp_inv, Fp, Kp, N, M);
	computeMd(Md, Fp, Qp_inv, Mp, N, M);

	hipFree(Gp_Qp_inv);
}

void computeTheta(float *theta, float *Qd, int N)
{
	float *Qdn = newMatrixCUDA(N,N);
	matrixNeg(Qdn, Qd, N, N);

	float *one = newMatrixCUDA(N,1);
	initMat(one, 1, N);

	float *tmp = newMatrixCUDA(N,1);
	matrixMultiply(tmp, Qdn, 0, one, 0, N,N,1);

	diagonalAdd(theta, tmp, N);

	hipFree(Qdn);
	hipFree(one);
	hipFree(tmp);
}

void computeQdp_theta(float *Qdp_theta, float *Qd, float *theta, int N)
{
	matrixPos(Qdp_theta, Qd, N, N);
	matrixAdd(Qdp_theta, theta, 1, N, N);
}

void computeQdn_theta(float *Qdn_theta, float *Qd, float *theta, int N)
{
	matrixNeg(Qdn_theta, Qd, N, N);
	matrixAdd(Qdn_theta, theta, 1, N, N);
}

void computealphaY(float *alphaY, float *ph, float *Qd, float *Y, float *Fd, int N)
{
	float *temp = newMatrixCUDA(1,N);

	matrixMultiply(temp, ph, 1, Qd, 0, 1, N, N);
	matrixMultiply(temp, temp, 0, ph, 0, 1, N, 1);

	float *com = newMatrix(1,1);
	copyToHost(com,temp,1,1);
	if(com[0] > 0)
	{
		float *temp2 = newMatrixCUDA(1,N);

		matrixMultiply(temp2, Y, 1, Qd, 0, 1, N, N);
		
		matrixAdd(temp2, Fd, 1, 1, N);
		
		matrixMultiply(temp2, temp2, 0, ph, 0, 1, N, 1);

		float *com2 = newMatrix(1,1);
		copyToHost(com2, temp2, 1,1);
		*alphaY = -com2[0]/com[0];

		free(com2);
		hipFree(temp2);
	}
	else
	{
		alphaY = 0;
	}
	free(com);
	hipFree(temp);
}

void updateY1(float *Y_next, float *Y, float alphaY, float *ph, int N)
{
	copyMatrix(Y_next, Y, N, 1);
	matrixAdd(Y_next, ph, alphaY, N, 1);
}

void updateY2(float *Y_next, float *Y, float *Qdp_theta, float *Qdn_theta, float *Fd, float *Fdp, float *Fdn, int N)
{
	float *numerator = newMatrixCUDA(N,1);
	float *denominator = newMatrixCUDA(N,1);

	matrixMultiply(numerator, Qdn_theta, 0, Y, 0, N, N, 1);
	matrixMultiply(denominator, Qdp_theta, 0, Y, 0, N, N, 1);

	matrixAdd(numerator, Fdn, 1, N, 1);
	matrixAdd(denominator, Fdp, 1, N, 1);

	updY(Y_next, numerator, denominator, Y, N);

	hipFree(numerator);
	hipFree(denominator);
}

void computeph(float *ph, float *Qd, float *Y, float *Fd, int N)
{
	matrixMultiply(ph, Qd, 0, Y, 0, N, N, 1);
	matrixAdd(ph, ph, 1, N, 1);
	matrixNeg(ph, ph, N, 1);
}

int checkFeas(float *U, float *Gp, float *Kp, int N, int M)
{
	float *tmp = newMatrixCUDA(N,1);
	matrixMultiply(tmp, Gp, 0, U, 0, N, M, 1);
	int re = 1;
	compare(tmp, Kp, &re, N);

	hipFree(tmp);
	return re;
}	

float computeCost(float *Z, float *Q, float *F, float *M, int N)
{
	float *J=newMatrixCUDA(1,1);

	float *tmp = newMatrixCUDA(1,N);
	matrixMultiply(tmp, Z, 1, Q, 0, 1, N, N);
	matrixMultiply(tmp, tmp, 0, Z, 0, 1, N, 1);

	matrixAdd(J, tmp, 0.5, 1,1);

	matrixMultiply(tmp, F, 1, Z, 0, 1, N, 1);

	matrixAdd(J, tmp, 1, 1,1);

	matrixAdd(J, M, 0.5, 1,1);
	

	float *hJ = newMatrix(1,1);
	copyToHost(hJ,J,1,1);

	float cost = hJ[0];
	free(hJ);
	hipFree(J);
	hipFree(tmp);

	return cost;
}

int terminate(float *Y, float *Qd, float *Fd, float *Md, float *U, float *Qp, float *Qp_inv, float *Fp, float *Mp, float *Gp, float *Kp, int N, int M)
{
	computeUfromY(U, Y, Fp, Gp, Qp_inv, N, M);

	if(!checkFeas(U, Gp, Kp, N, M))	return 0;

	float Jd = computeCost(Y, Qd, Fd, Md, N);
	float Jp = computeCost(U, Qp, Fp, Mp, M);

	if(Jp>-Jd)	return 0;
	if(Jp+Jd>eaj)	return 0;
	if((Jp+Jd)/fabs(Jd)>erj) return 0;

	return 1;
}

void solveQuadraticDual(float *Y, float *Qd, float *Fd, float *Md, float *U, float *Qp, float *Qp_inv, float *Fp, float *Mp, float *Gp, float *Kp, int N, int M)
{
	float *theta = newMatrixCUDA(N,N);
	float *Qdp_theta = newMatrixCUDA(N,N);
	float *Qdn_theta = newMatrixCUDA(N,N);
	float *Y_next = newMatrixCUDA(N,1);
	
	float *Fdn = newMatrixCUDA(N,1);
	float *Fdp = newMatrixCUDA(N,1);

	matrixPos(Fdp, Fd, N, 1);
	matrixNeg(Fdn, Fd, N, 1);
	
	computeTheta(theta, Qd, N);
	computeQdp_theta(Qdp_theta, Qd, theta, N);
	computeQdn_theta(Qdn_theta, Qd, theta, N);

	initMat(Y, 1000.0, N);

	float *ph = newMatrixCUDA(N,1);
	long int h=1;

	while(h<NUM_ITER)
	//while(!terminate(Y, Qd, Fd, Md, U, Qp, Qp_inv, Fp, Mp, Gp, Kp, N, M))
	{	
		if(1)
		{
			//update
			updateY2(Y_next, Y, Qdp_theta, Qdn_theta, Fd, Fdp, Fdn, N);			
		}
//		else
//		{
//			// accelerate
//			float alphaY=0;
//			computeph(ph, Qd, Y, Fd, N);
//			computealphaY(&alphaY, ph, Qd, Y, Fd, N);
//			updateY1(Y_next, Y, alphaY/10, ph, N);
//
//		}

		copyMatrix(Y, Y_next, N, 1);

		h++;
	}
	printf("Printing number of iterations = %ld\n",h);

	hipFree(theta);
	hipFree(Qdp_theta);
	hipFree(Qdn_theta);
	hipFree(Y_next);
	hipFree(ph);
	hipFree(Fdp);
	hipFree(Fdn);
}

void input(float *Qp_inv, float *Fp, float *Mp, float *Gp, float *Kp, float *x, float *D, float *theta, float *Z, int N, int M, char *fi)
{
	int tmp;
	FILE *fp = fopen(fi, "r");
	fscanf(fp, "%d%d", &tmp,&tmp);
	for(int i=0;i<M;i++)
	{
		fscanf(fp, "%f", &Qp_inv[i*M+i]);
	}

	for(int i=0;i<M;i++)
	{
		fscanf(fp, "%f", &Fp[i]);
	}

 	fscanf(fp, "%f", Mp);

	for(int i=0;i<N;i++)
	{
		fscanf(fp, "%f", &Kp[i]);
	}

	for(int i=0;i<N;i++)
	{
		Kp[i] = fabs(10.0*rand()/RAND_MAX);
		for(int j=0;j<M;j++)
		{
			int tmp;
			fscanf(fp, "%d",&tmp);
			if(tmp%3 == 0)
			{
				Gp[i*M+j] = 0;
			}
			else if(tmp%3==2)
			{
				Gp[i*M+j] = -1;
			}
			else
			{
				Gp[i*M+j] = 1;
			}
		}
	}
}

int main(int argc, char *argv[])
{
	// QP is of parametric from 
	// J(U) = min U 1/2*U'QpU + Fp'U + 1/2*Mp
	// st GpU <= Kp
	
	hipDeviceReset();
	 
	int N, M;
	FILE *fp;
	fp = fopen(argv[1], "r");
	fscanf(fp, "%d%d", &M, &N);
	fclose(fp);
	// host matrix
	float *hQp_inv = newMatrix(M,M);
	float *hQp = newMatrix(M,M);

	float *hFp1;
	float *hFp2;
	float *hFp3;

	float *hMp1;
	float *hMp2;
	float *hMp3;
	float *hMp4;
	float *hMp5;
	float *hMp6;

	float *hFp = newMatrix(M,1);
	float *hMp = newMatrix(1,1);
	float *hGp;
	float *hKp;
	float *hx;
	float *hD; 
	float *htheta; 
	float *hZ; 

	hFp1 = newMatrix(nInput*pHorizon, nDis*pHorizon);
	hFp2 = newMatrix(nInput*pHorizon, nState);
	hFp3 = newMatrix(1, nInput*pHorizon);
	hMp1 = newMatrix(nState, nState);
	hMp2 = newMatrix(nDis*pHorizon, nState);
	hMp3 = newMatrix(nDis*pHorizon, nDis*pHorizon);
	hMp4 = newMatrix(1, nState);
	hMp5 = newMatrix(1, nDis*pHorizon);
	hMp6 = newMatrix(1,1);
	hGp = newMatrix(N,M);
	hKp = newMatrix(N,1);
	hZ = newMatrix(nOutput*pHorizon, nState);
	htheta = newMatrix(nOutput*pHorizon, nDis*pHorizon);
	hD = newMatrix(nDis*pHorizon,1);
	hx = newMatrix(nState, 1);

	// device matrix
	float *Qp_inv = newMatrixCUDA(M,M);
	float *Qp = newMatrixCUDA(M,M);

	float *Fp1;
	float *Fp2;
	float *Fp3;

	float *Mp1;
	float *Mp2;
	float *Mp3;
	float *Mp4;
	float *Mp5;
	float *Mp6;

	float *Fp = newMatrixCUDA(M,1);
	float *Mp = newMatrixCUDA(1,1);
	float *Gp;
	float *Kp;
	float *x;
	float *D; 
	float *theta; 
	float *Z; 

	Fp1 = newMatrixCUDA(nInput*pHorizon, nDis*pHorizon);
	Fp2 = newMatrixCUDA(nInput*pHorizon, nState);
	Fp3 = newMatrixCUDA(1, nInput*pHorizon);
	Mp1 = newMatrixCUDA(nState, nState);
	Mp2 = newMatrixCUDA(nDis*pHorizon, nState);
	Mp3 = newMatrixCUDA(nDis*pHorizon, nDis*pHorizon);
	Mp4 = newMatrixCUDA(1, nState);
	Mp5 = newMatrixCUDA(1, nDis*pHorizon);
	Mp6 = newMatrixCUDA(1,1);
	Gp = newMatrixCUDA(N,M);
	Kp = newMatrixCUDA(N,1);
	Z = newMatrixCUDA(nOutput*pHorizon, nState);
	theta = newMatrixCUDA(nOutput*pHorizon, nDis*pHorizon);
	D = newMatrixCUDA(nDis*pHorizon,1);
	x = newMatrixCUDA(nState, 1);	

	input(hQp_inv, hFp, hMp, hGp, hKp, hx, hD, htheta, hZ, N, M, argv[1]);
	float tmp;
	fscanf(fp, "%f",&tmp);
	fclose(fp);
	Gauss_Jordan(hQp_inv, hQp, M);
	copyToDevice(Qp_inv, hQp_inv, M, M);
	copyToDevice(Qp, hQp, M, M);
	copyToDevice(Fp1, hFp1, nInput*pHorizon, nDis*pHorizon);
	copyToDevice(Fp2, hFp2, nInput*pHorizon, nState);
	copyToDevice(Fp3, hFp3, 1, nInput*pHorizon);
	copyToDevice(Mp1, hMp1, nState, nState);
	copyToDevice(Mp2, hMp2, nDis*pHorizon, nState);
	copyToDevice(Mp3, hMp3, nDis*pHorizon, nDis*pHorizon);
	copyToDevice(Mp4, hMp4, 1, nState);
	copyToDevice(Mp5, hMp5, 1, nDis*pHorizon);
	copyToDevice(Mp6, hMp6, 1,1);
	copyToDevice(Gp, hGp, 4*pHorizon*nInput, nInput*pHorizon);
	copyToDevice(Kp, hKp, 1,4*pHorizon*nInput);
	copyToDevice(Z, hZ, nOutput*pHorizon, nState);
	copyToDevice(D, hD, nDis*pHorizon,1);
	copyToDevice(theta, htheta, nOutput*pHorizon, nDis*pHorizon);
	copyToDevice(x, hx, nState, 1);

	//computeFp(Fp, Fp1, Fp2, Fp3, D, x);
	//computeMp(Mp, Mp1, Mp2, Mp3, Mp4, Mp5, Mp6, D, x);

	// matrices and vectors required for dual form of QP
	float *Qd = newMatrixCUDA(N,N);
	float *Fd = newMatrixCUDA(N,1);
	float *Md = newMatrixCUDA(1,1);	
	float *Y  = newMatrixCUDA(N,1);
	float *U  = newMatrixCUDA(M,1);

	convertToDual(Qd, Fd, Md, Qp_inv, Gp, Kp, Fp, Mp, N, M);

	solveQuadraticDual(Y, Qd, Fd, Md, U, Qp, Qp_inv, Fp, Mp, Gp, Kp, N, M);

	computeUfromY(U, Y, Fp, Gp, Qp_inv, N, M);

	float Jp = computeCost(U, Qp, Fp, Mp, M);
	float Jd = computeCost(Y, Qd, Fd, Md, N);

	printf("Jp = %f\n", Jp);
	printf("Jd = %f\n", Jd);
	
	float *hU = newMatrix(M,1);
	float *hY = newMatrix(N,1);

	copyToHost(hU,U,M,1);
	copyToHost(hY,Y,N,1);

	printf("Printing U*\n");
	for(int i=0;i<M;i++)
	{
		printf("\t%f\n", hU[i]);
	}

	free(hQp_inv);
	free(hQp);
	free(hFp1);
	free(hFp2);
	free(hFp3);
	free(hMp1);
	free(hMp2);
	free(hMp3);
	free(hMp4);
	free(hMp5);
	free(hMp6); 
	free(hFp);
	free(hMp);
	free(hGp);
	free(hKp);
	free(hx);
	free(hD);
	free(htheta);
	free(hZ);

	hipFree(Qp_inv);
	hipFree(Qp);
	hipFree(Fp1);
	hipFree(Fp2);
	hipFree(Fp3);
	hipFree(Mp1);
	hipFree(Mp2);
	hipFree(Mp3);
	hipFree(Mp4);
	hipFree(Mp5);
	hipFree(Mp6); 
	hipFree(Fp);
	hipFree(Mp);
	hipFree(Gp);
	hipFree(Kp);
	hipFree(x);
	hipFree(D);
	hipFree(theta);
	hipFree(Z);
	
	hipFree(Qd);
	hipFree(Fd);
	hipFree(Md);
	hipFree(Y);
	hipFree(U);
}