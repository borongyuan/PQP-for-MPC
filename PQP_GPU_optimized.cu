/**************************************************************************
* This file contains implementation of pqp (parallel quadratic programming)
* GPU version optimised with TILE and shared memory for MPC Term Project of HP3 Course.
* Group 7 CSE Dept. IIT KGP
*	Objective function: 1/2 U'QpU + Fp'U + 1/2 Mp
*	Constraints: GpU <= Kp
**************************************************************************/

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>


#define NUM_ITER 1000

#define pHorizon 1
#define nState 29
#define nInput 7
#define nOutput 7
#define nDis 1

#define erc 1e-6
#define eac 1e-6
#define eaj 1e-6
#define erj 1e-6

#define TILE_DIM 32
#define BLOCK_ROWS 8
#define BLOCK_SIZE 16
#define BLK_ROWS 32
#define BLK_COLS 32
//size of the share memory tile in the device
#define TILE_SIZE BLK_ROWS

__global__ void printMat(float *mat, int N, int M)
{
	printf("printing mat\n");
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<M;j++)
		{
			printf("%f ",mat[i*M+j]);
		}
		printf("\n");
	}
	printf("\n");
}



__global__ void initMatCuda(float *mat, float val, int N)								
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<N)	
	{
		mat[id] = val;
	}
}

/**************************************************************************
* This is utility function initialize the matrix
*   1. Parameter is float type matrix pointer (*mat), float val, 
*		size of matrix 
*   2. Return type void
**************************************************************************/
void initMat(float *mat, float val, int N)								
{
	dim3 block = 1024;
	dim3 grid = (N+1024-1)/1024;

	initMatCuda<<<grid, block>>>(mat, val, N);
}



float *newMatrixCUDA(int n, int m)			
{
	float *tmp = NULL;
	
	hipError_t err = hipMalloc((void **)&tmp, n*m*sizeof(float));

	if ( err != hipSuccess )
	{
		printf (" Failed to allocate device matrix! %s\n", hipGetErrorString(err));
		exit ( EXIT_FAILURE ) ;
	}

	initMat(tmp, 0, n*m);
	return tmp;
}

/**************************************************************************
* This is utility function for create new  matrix
*   1. Parameter is (int n, int m) dimension of (n X m matrix) , 
*	2. Return pointer of new matrix
*   3. This function create dynamic size matrix using malloc
**************************************************************************/
float *newMatrix(int n, int m)			
{
	float *tmp = (float *)malloc(n*m*sizeof(float));
	for(int i=0;i<n*m;i++)
	{
		tmp[i] = 0;
	}
	return tmp;
}



void copyToDevice(float *dM, float *hM, int n, int m)
{
	int size = n*m;
	hipMemcpy (dM ,hM, size * sizeof ( float ) , hipMemcpyHostToDevice );
}

void copyToHost(float *hM, float *dM, int n, int m)
{
	int size = n*m;
	hipMemcpy (hM ,dM, size * sizeof ( float ) , hipMemcpyDeviceToHost );
}



__global__ void copyMatrixCuda(float *output, float *mat, int a, int b)		
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<a*b)	
	{
		output[id] = mat[id];
	}
}

void copyMatrix(float *output, float *mat, int a, int b)		
{
	dim3 block = 1024;
	dim3 grid = (a*b+1024-1)/1024;

	copyMatrixCuda<<<grid,block>>>(output, mat, a, b);
}



__global__ void transposeCuda(float *odata, float *idata, int n, int m)
{
  __shared__ float tile[TILE_DIM][TILE_DIM+1];
    
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  //int width = gridDim.x * TILE_DIM;

  for(int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
  {
	if(x<m && y<n)
	{
     tile[threadIdx.x][threadIdx.y] = idata[y*m+x];
	}
  }
 
  __syncthreads();
	
  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for(int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
  {
	if(y<m && x<n){
     odata[(y*n) + x] = tile[threadIdx.y][threadIdx.x];
	 }
  }
  
}



void transpose(float *odata, float *idata, int n, int m)
{
	dim3 grid((n+TILE_DIM-1)/TILE_DIM, (m+TILE_DIM-1)/TILE_DIM, 1);
	dim3 block(TILE_DIM, TILE_DIM, 1);

	
	transposeCuda<<<grid,block>>>(odata,idata,n,m);
}

__global__ void matrixMultiplyCuda(float *output, float *matrix1, float *matrix2, int a, int b, int c)
{
	//declare shared memory matrices for matrix1 and matrix2 matrices
	__shared__ float shared_mat1_tile[TILE_SIZE][TILE_SIZE];
	__shared__ float shared_mat2_tile[TILE_SIZE][TILE_SIZE];
	int tsize;
	if(a!=1 && c!=1){	
	tsize=TILE_SIZE;
	}
	else{		
	tsize=1;
	}
	
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	//check if thread directly maps to the dimensions of the resulting matrix
	if (row < a && col < c)
	{
		float result = 0.0;
		int k;
		int phase;
		
		//calculate output matrix indexes in phases. Each phase shares 
		//TILE_SIZE * TILE_SIZE data copied to the shared matrix mat1 
		//and matrix mat2.
		for (phase = 0; phase <= b/tsize; phase++)
		{
			if(phase*tsize+tx < b)
				shared_mat1_tile[ty][tx] = matrix1[row * b + phase * tsize + tx];
			else
				shared_mat1_tile[ty][tx] = 0;
			if(phase*tsize+ty < b)
				shared_mat2_tile[ty][tx] = matrix2[(phase * tsize + ty) * c + col];
			else
				shared_mat2_tile[ty][tx] = 0;

			__syncthreads();
			
			for (k = 0; k < tsize; k++)
			{
				if (k + (phase * tsize) < b) 
				{
					result += (shared_mat1_tile[ty][k] * shared_mat2_tile[k][tx]);
				}
			}
			__syncthreads();
		}	
		output[row * c + col] = result;
	}
}


void matrixMultiply(float *output, float *mat1, int transpose1, float *mat2, int transpose2, int a, int b, int c) 		//mat1-a*b	mat2-b*c 	
{
	float *tmp = newMatrixCUDA(a,c);
	
	float *matrix1;
	float *matrix2;
	
	if(transpose1 && a!=1 && b!=1)
	{
		matrix1 = newMatrixCUDA(a,b);
		transpose(matrix1, mat1, b,a);
			
	}
	else
	{
		matrix1 = mat1;
	}
	
	if(transpose2 && b!=1 && c!=1)
	{
		matrix2 = newMatrixCUDA(b,c);
		transpose(matrix2, mat1, c,b);
		
	}
	else
	{
		matrix2 = mat2;
	}
	int B_C, B_R;
	if(a!=1 && c!=1)
	{
		B_C=BLK_COLS;
		B_R=BLK_ROWS;
	}
	else{
		B_C=1;
		B_R=1;
	}
	dim3 block(B_C,B_R);
	dim3 grid((c+B_C-1)/B_C,(a+B_R-1)/B_R);
	matrixMultiplyCuda<<<grid, block>>>(output, matrix1, matrix2, a, b, c);
	
	if(transpose1 && a!=1 && b!=1)
	{
		hipFree(matrix1);
	}
	if(transpose2 && b!=1 && c!=1)
	{
		hipFree(matrix2);
	}

	hipFree(tmp);
}

/**************************************************************************
* This is utility function for generating addition or substraction 
*	of two matrix
*   1. Parameter is (pointer of matrix1, pointer of matrix2, float sign,int n int m)
*		dimension of (n X m matrix) 
*	2. sign parameters for decide addition or substraction
*	3. Result write back in matrix1
**************************************************************************/

__global__ void matrixAddCuda(float *A, float *B, float sign, int a, int b) 			// adds b to a 	
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<a*b)	
	{
		A[id] += sign * B[id];
	}
}

void matrixAdd(float *A, float *B, float sign, int a, int b) 			// adds b to a 	
{
	dim3 block = 1024;
	dim3 grid = (a*b+1024-1)/1024;

	matrixAddCuda<<<grid,block>>>(A,B,sign,a,b);
}



__global__ void negateMatrixCuda(float *mat, int n, int m)			
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<n*m)	
	{
		mat[id] = -mat[id];
	}
}

void negateMatrix(float *mat, int n, int m)			
{
	dim3 block = 1024;
	dim3 grid = (n*m+1024-1)/1024;

	negateMatrixCuda<<<grid,block>>>(mat,n,m);
}



__global__ void matrixPosCuda(float *mat1, float *mat2, int n, int m)			
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<n*m)	
	{
		mat1[id] = fmaxf(0.0, mat2[id]);

	}
}

void matrixPos(float *mat1, float *mat2, int n, int m)			
{
	dim3 block = 1024;
	dim3 grid = (n*m+1024-1)/1024;

	matrixPosCuda<<<grid,block>>>(mat1,mat2,n,m);
}



__global__ void matrixNegCuda(float *mat1, float *mat2, int n, int m)			
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<n*m)	
	{
		mat1[id] = fmaxf(0.0, -mat2[id]);
	}
}

void matrixNeg(float *mat1, float *mat2, int n, int m)			
{
	dim3 block = 1024;
	dim3 grid = (n*m+1024-1)/1024;

	matrixNegCuda<<<grid,block>>>(mat1,mat2,n,m);
}



__global__ void diagonalAddCuda(float *theta, float *tmp, int N)			
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<N)	
	{
		// printf("tmp %f\n",tmp[i]);
		theta[id*N+id] = fmaxf(tmp[id],5.0);
	}
}

void diagonalAdd(float *theta, float *tmp, int N)			
{
	dim3 block = 1024;
	dim3 grid = (N+1024-1)/1024;

	diagonalAddCuda<<<grid,block>>>(theta,tmp,N);
}



__global__ void compareCuda(float *GpU, float *Kp, int *re, int N)				
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	if(id<N)	
	{
		if(GpU[id] > Kp[id]+fmaxf(erc*Kp[id], eac))
		{
			*re = 0;
		}
	}
}

void compare(float *GpU, float *Kp, int *re, int N)				
{
	dim3 block = 1024;
	dim3 grid = (N+1024-1)/1024;

	compareCuda<<<grid,block>>>(GpU, Kp, re, N);
}




__global__ void updYCuda(float *Y_next, float *numerator, float *denominator, float *Y, int N)   
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int id = blockNum * (blockDim.x * blockDim.y * blockDim.z) + threadNum;
	
	if(id<N)
	{
		Y_next[id] = numerator[id]/denominator[id]*Y[id];
	}
}

void updY(float *Y_next, float *numerator, float *denominator, float *Y, int N)   
{
	dim3 block = 1024;
	dim3 grid = (N+1023)/1024;
	
	updYCuda<<<grid, block>>>(Y_next, numerator, denominator, Y, N);
}




void Gauss_Jordan(float *A,float *res, int N)
{
    /*
    size=Size of input matrix
    A=input matrix
    res= inverted matrix
    */
    float temp;
    float *matrix = newMatrix(N, 2*N);

    for (int i = 0; i < N; i++) 
    { 
        for (int j = 0; j < 2 * N; j++) 
        { 
            matrix[i*2*N+j]=0;
            if (j == (i + N)) 
                matrix[i*2*N+j] = 1; 
        } 
    }

    for (int i = 0; i < N; i++) 
    { 
        for (int j = 0; j < N; j++) 
        { 
            matrix[i*2*N+j]=A[i*N+j];

        } 
    }

    for (int i = N - 1; i > 0; i--) 
    { 
        if (matrix[(i - 1)*2*N+0] < matrix[i*2*N+0]) 
            for (int j = 0; j < 2 * N; j++) 
            { 
                temp = matrix[i*2*N+j]; 
                matrix[i*2*N+j] = matrix[(i - 1)*2*N+j]; 
                matrix[(i - 1)*2*N+j] = temp; 
            } 
    }

    for (int i = 0; i < N; i++)
    { 

        for (int j = 0; j < N; j++) 
        { 
            if (j != i) 
            { 
                temp = matrix[j*2*N+i] / matrix[i*2*N+i]; 
                for (int k = 0; k < 2 * N; k++) 
                { 
                    matrix[j*2*N+k] -= matrix[i*2*N+k] * temp; 
                } 
            } 
        } 
    } 

    for (int i = 0; i < N; i++)
    { 
        temp = matrix[i*2*N+i]; 
        for (int j = 0; j < 2 * N; j++)
        {
            matrix[i*2*N+j] = matrix[i*2*N+j] / temp; 
        } 
    }

    for (int i = 0; i < N; i++) 
    { 
        for (int j = N; j <2*N; j++) 
        { 
            res[i*N+j-N]=matrix[i*2*N+j];

        } 
    }

    free(matrix);
}



void computeUfromY(float *U, float *Y, float *Fp, float *Gp, float *Qp_inv, int N, int M)
{
	float *tmp = newMatrixCUDA(M,1);
	matrixMultiply(tmp, Gp, 1, Y, 0, M, N, 1);
	matrixAdd(tmp, Fp, 1, M, 1);
	matrixMultiply(U, Qp_inv, 0, tmp, 0, M, M, 1);
	negateMatrix(U, M, 1);
	hipFree(tmp);
}

void computeFp(float *Fp, float *Fp1, float *Fp2, float *Fp3, float *D, float *x)
{
	matrixMultiply(Fp, Fp1, 0, D, 0, nInput*pHorizon, nDis*pHorizon, 1);
	float *Fp2x = newMatrixCUDA(nInput*pHorizon,1);
	matrixMultiply(Fp2x, Fp2, 0, x, 0, nInput*pHorizon, nState, 1);
	matrixAdd(Fp, Fp2x, 1, nInput*pHorizon, 1);
	matrixAdd(Fp, Fp3, -1, nInput*pHorizon, 1);
	
	hipFree(Fp2x);	
}

void computeMp(float *Mp, float *Mp1, float *Mp2, float *Mp3, float *Mp4, float *Mp5, float *Mp6, float *D, float *x)
{
	initMat(Mp, 0, 1);

	float *tmp = newMatrixCUDA(1,nState);
	matrixMultiply(tmp, x, 1, Mp1, 0, 1, nState, nState);
	matrixMultiply(tmp, tmp, 0, x, 0, 1, nState, 1);

	matrixAdd(Mp, tmp, 0.5, 1,1);

	matrixMultiply(tmp, D, 1, Mp2, 0, 1, nDis*pHorizon, nState);
	matrixMultiply(tmp, tmp, 0, x, 0, 1, nState, 1);

	matrixAdd(Mp, tmp, 0.5, 1,1);
	
	matrixMultiply(tmp, Mp4, 1, x, 0, 1, nState, 1);

	matrixAdd(Mp, tmp, 0.5, 1,1);

	hipFree(tmp);
	tmp = newMatrixCUDA(1, nDis*pHorizon);
	matrixMultiply(tmp, D, 1, Mp3, 0, 1, nDis*pHorizon, nDis*pHorizon);
	matrixMultiply(tmp, tmp, 0, D, 0, 1, nDis*pHorizon, 1);

	matrixAdd(Mp, tmp, 0.5, 1,1);

	matrixMultiply(tmp, Mp5, 1, D, 0, 1, nDis*pHorizon, 1);

	matrixAdd(Mp, tmp, 0.5, 1,1);

	matrixAdd(Mp, Mp6, 0.5, 1,1);
	hipFree(tmp);
}

void computeQd(float *Qd, float *Gp_Qp_inv, float *Gp, int N, int M)
{
	matrixMultiply(Qd, Gp_Qp_inv, 0, Gp, 1, N, M, N);	
}

void computeFd(float *Fd, float *Gp_Qp_inv, float *Fp, float *Kp, int N, int M)
{
	matrixMultiply(Fd, Gp_Qp_inv, 0, Fp, 0, N, M, 1);
	matrixAdd(Fd, Kp, 1, N, 1);
}

void computeMd(float *Md, float *Fp, float* Qp_inv, float* Mp, int N, int M)
{
	float *tmp = newMatrixCUDA(1,M);
	matrixMultiply(tmp, Fp, 1, Qp_inv, 0, 1, M, M);
	matrixMultiply(Md, tmp, 0, Fp, 0, 1, M, 1);
	matrixAdd(Md, Mp, -1, 1, 1);
	hipFree(tmp);
}

void convertToDual(float *Qd, float *Fd, float *Md, float *Qp_inv, float *Gp, float *Kp, float *Fp, float *Mp, int N, int M)
{	
	float *Gp_Qp_inv = newMatrixCUDA(N,M);
	matrixMultiply(Gp_Qp_inv, Gp, 0, Qp_inv, 0, N, M, M);
	computeQd(Qd, Gp_Qp_inv, Gp, N, M);
	computeFd(Fd, Gp_Qp_inv, Fp, Kp, N, M);
	computeMd(Md, Fp, Qp_inv, Mp, N, M);

	hipFree(Gp_Qp_inv);
}

void computeTheta(float *theta, float *Qd, int N)
{
	float *Qdn = newMatrixCUDA(N,N);
	matrixNeg(Qdn, Qd, N, N);

	float *one = newMatrixCUDA(N,1);
	initMat(one, 1, N);

	float *tmp = newMatrixCUDA(N,1);
	matrixMultiply(tmp, Qdn, 0, one, 0, N,N,1);

	diagonalAdd(theta, tmp, N);

	hipFree(Qdn);
	hipFree(one);
	hipFree(tmp);
}

void computeQdp_theta(float *Qdp_theta, float *Qd, float *theta, int N)
{
	matrixPos(Qdp_theta, Qd, N, N);
	matrixAdd(Qdp_theta, theta, 1, N, N);
}

void computeQdn_theta(float *Qdn_theta, float *Qd, float *theta, int N)
{
	matrixNeg(Qdn_theta, Qd, N, N);
	matrixAdd(Qdn_theta, theta, 1, N, N);
}

void computealphaY(float *alphaY, float *ph, float *Qd, float *Y, float *Fd, int N)
{
	float *temp = newMatrixCUDA(1,N);

	matrixMultiply(temp, ph, 1, Qd, 0, 1, N, N);
	matrixMultiply(temp, temp, 0, ph, 0, 1, N, 1);

	float *com = newMatrix(1,1);
	copyToHost(com,temp,1,1);
	if(com[0] > 0)
	{
		float *temp2 = newMatrixCUDA(1,N);

		matrixMultiply(temp2, Y, 1, Qd, 0, 1, N, N);
		
		matrixAdd(temp2, Fd, 1, 1, N);
		
		matrixMultiply(temp2, temp2, 0, ph, 0, 1, N, 1);

		float *com2 = newMatrix(1,1);
		copyToHost(com2, temp2, 1,1);
		*alphaY = -com2[0]/com[0];

		free(com2);
		hipFree(temp2);
	}
	else
	{
		alphaY = 0;
	}
	free(com);
	hipFree(temp);
}

void updateY1(float *Y_next, float *Y, float alphaY, float *ph, int N)
{
	copyMatrix(Y_next, Y, N, 1);
	matrixAdd(Y_next, ph, alphaY, N, 1);
}

void updateY2(float *Y_next, float *Y, float *Qdp_theta, float *Qdn_theta, float *Fd, float *Fdp, float *Fdn, int N)
{
	float *numerator = newMatrixCUDA(N,1);
	float *denominator = newMatrixCUDA(N,1);

	matrixMultiply(numerator, Qdn_theta, 0, Y, 0, N, N, 1);
	matrixMultiply(denominator, Qdp_theta, 0, Y, 0, N, N, 1);

	matrixAdd(numerator, Fdn, 1, N, 1);
	matrixAdd(denominator, Fdp, 1, N, 1);

	updY(Y_next, numerator, denominator, Y, N);

	hipFree(numerator);
	hipFree(denominator);
}

void computeph(float *ph, float *Qd, float *Y, float *Fd, int N)
{
	matrixMultiply(ph, Qd, 0, Y, 0, N, N, 1);
	matrixAdd(ph, ph, 1, N, 1);
	matrixNeg(ph, ph, N, 1);
}

int checkFeas(float *U, float *Gp, float *Kp, int N, int M)
{
	float *tmp = newMatrixCUDA(N,1);
	matrixMultiply(tmp, Gp, 0, U, 0, N, M, 1);
	int re = 1;
	compare(tmp, Kp, &re, N);

	hipFree(tmp);
	return re;
}	

float computeCost(float *Z, float *Q, float *F, float *M, int N)
{
	float *J=newMatrixCUDA(1,1);

	float *tmp = newMatrixCUDA(1,N);
	matrixMultiply(tmp, Z, 1, Q, 0, 1, N, N);
	matrixMultiply(tmp, tmp, 0, Z, 0, 1, N, 1);

	matrixAdd(J, tmp, 0.5, 1,1);

	matrixMultiply(tmp, F, 1, Z, 0, 1, N, 1);

	matrixAdd(J, tmp, 1, 1,1);

	matrixAdd(J, M, 0.5, 1,1);
	

	float *hJ = newMatrix(1,1);
	copyToHost(hJ,J,1,1);

	float cost = hJ[0];
	free(hJ);
	hipFree(J);
	hipFree(tmp);

	return cost;
}

int terminate(float *Y, float *Qd, float *Fd, float *Md, float *U, float *Qp, float *Qp_inv, float *Fp, float *Mp, float *Gp, float *Kp, int N, int M)
{
	computeUfromY(U, Y, Fp, Gp, Qp_inv, N, M);

	if(!checkFeas(U, Gp, Kp, N, M))	return 0;

	float Jd = computeCost(Y, Qd, Fd, Md, N);
	float Jp = computeCost(U, Qp, Fp, Mp, M);

	if(Jp>-Jd)	return 0;
	if(Jp+Jd>eaj)	return 0;
	if((Jp+Jd)/fabs(Jd)>erj) return 0;

	return 1;
}

void solveQuadraticDual(float *Y, float *Qd, float *Fd, float *Md, float *U, float *Qp, float *Qp_inv, float *Fp, float *Mp, float *Gp, float *Kp, int N, int M)
{
	float *theta = newMatrixCUDA(N,N);
	float *Qdp_theta = newMatrixCUDA(N,N);
	float *Qdn_theta = newMatrixCUDA(N,N);
	float *Y_next = newMatrixCUDA(N,1);
	
	float *Fdn = newMatrixCUDA(N,1);
	float *Fdp = newMatrixCUDA(N,1);

	matrixPos(Fdp, Fd, N, 1);
	matrixNeg(Fdn, Fd, N, 1);
	
	computeTheta(theta, Qd, N);
	computeQdp_theta(Qdp_theta, Qd, theta, N);
	computeQdn_theta(Qdn_theta, Qd, theta, N);

	initMat(Y, 1000.0, N);
	// for(int i=0;i<N;i++) Y[i] = i+1;

	float *ph = newMatrixCUDA(N,1);
	long int h=1;

	while(!terminate(Y, Qd, Fd, Md, U, Qp, Qp_inv, Fp, Mp, Gp, Kp, N, M))
	{	
		if(1)
		{
			//update
			updateY2(Y_next, Y, Qdp_theta, Qdn_theta, Fd, Fdp, Fdn, N);			
		}
//		else
//		{
//			// accelerate
//			float alphaY=0;
//			computeph(ph, Qd, Y, Fd, N);
//			computealphaY(&alphaY, ph, Qd, Y, Fd, N);
//			updateY1(Y_next, Y, alphaY/10, ph, N);
//
//		}

		copyMatrix(Y, Y_next, N, 1);
		h++;
	}
	printf("Printing number of iterations = %ld\n",h);

	hipFree(theta);
	hipFree(Qdp_theta);
	hipFree(Qdn_theta);
	hipFree(Y_next);
	hipFree(ph);
	hipFree(Fdp);
	hipFree(Fdn);
}

void input(float* qp_inv, float* Fp1, float* Fp2, float * Fp3, float * Mp1, float * Mp2, float * Mp3, float* Mp4, float* Mp5, float* Mp6, float* Gp, float* Kp, float* x, float* D, float* theta, float* Z)
{
	FILE *fptr;
	int i,j;
	float num;

	//Fill Qp_inverse	
	fptr = fopen("./example/Qp_inv.txt","r");
	for(i=0;i<pHorizon*nInput;i++)
	{
		for(j=0;j<pHorizon*nInput;j++)
		{
			fscanf(fptr,"%f", &num);
			qp_inv[j*pHorizon*nInput+i] = num;
		}
	}
	fclose(fptr);

	//Fill Fp1
	fptr = fopen("./example/Fp1.txt","r");
	for(i=0;i<nDis*pHorizon;i++)
	{
		for(j=0;j<nInput*pHorizon;j++)
		{
			fscanf(fptr,"%f", &num);
			Fp1[j*nDis*pHorizon+i] = num;
		}
	}
	fclose(fptr);

	//Fill Fp2
	fptr = fopen("./example/Fp2.txt","r");
	for(i=0;i<nState;i++)
	{
		for(j=0;j<nInput*pHorizon;j++)
		{
			fscanf(fptr,"%f", &num);
			Fp2[j*nState+i] = num;
		}
	}
	fclose(fptr);

	//Fill Fp3
	fptr = fopen("./example/Fp3.txt","r");
	for(j=0;j<nInput*pHorizon;j++)
	{
		fscanf(fptr,"%f", &num);
		Fp3[j] = num;
	}
	fclose(fptr);

	//Fill Mp1
	fptr = fopen("./example/Mp1.txt","r");
	for(i=0;i<nState;i++)
	{
		for(j=0;j<nState;j++)
		{
			fscanf(fptr,"%f", &num);
			Mp1[j*nState+i] = num;
		}
	}
	fclose(fptr);

	//Fill Mp2
	fptr = fopen("./example/Mp2.txt","r");
	for(i=0;i<nState;i++)
	{
		for(j=0;j<nDis*pHorizon;j++)
		{
			fscanf(fptr,"%f", &num);
			Mp2[j*nState+i] = num;
		}
	}
	fclose(fptr);

	//Fill Mp3
	fptr = fopen("./example/Mp3.txt","r");
	for(i=0;i<nDis*pHorizon;i++)
	{
		for(j=0;j<nDis*pHorizon;j++)
		{
			fscanf(fptr,"%f", &num);
			Mp3[j*nDis*pHorizon+i] = num;
		}
	}
	fclose(fptr);

	//Fill Mp4
	fptr = fopen("./example/Mp4.txt","r");
	for(i=0;i<nState;i++)
	{
		fscanf(fptr,"%f", &num);
		Mp4[i] = num;
	}
	fclose(fptr);

	//Fill Mp5
	fptr = fopen("./example/Mp5.txt","r");
	for(i=0;i<nDis*pHorizon;i++)
	{
		fscanf(fptr,"%f", &num);
		Mp5[i] = num;
	}
	fclose(fptr);

	//Fill Mp6
	fptr = fopen("./example/Mp6.txt","r");
	fscanf(fptr,"%f", &num);
	Mp6[0] = num;
	fclose(fptr);

	//Fill Gp
	fptr = fopen("./example/Gp.txt","r");
	for(i=0;i<pHorizon*nInput;i++)
	{
		for(j=0;j<4*pHorizon*nInput;j++)
		{
			fscanf(fptr,"%f", &num);
			Gp[j*pHorizon*nInput+i] = num;
		}
	}
	fclose(fptr);

	//Fill Kp
	fptr = fopen("./example/Kp.txt","r");
	for(i=0;i<4*pHorizon*nInput;i++)
	{
		fscanf(fptr,"%f", &num);
		Kp[i] = num;
	}
	fclose(fptr);

	//Fill Z
	fptr = fopen("./example/Z.txt","r");
	for(i=0;i<nState;i++)
	{
		for(j=0;j<nOutput*pHorizon;j++)
		{
			fscanf(fptr,"%f", &num);
			Z[j*nState+i] = num;
		}
	}
	fclose(fptr);

	//Fill Theta
	fptr = fopen("./example/Theta.txt","r");
	for(i=0;i<nDis*pHorizon;i++)
	{
		for(j=0;j<nOutput*pHorizon;j++)
		{
			fscanf(fptr,"%f", &num);
			theta[j*nDis*pHorizon+i] = num;
		}
	}
	fclose(fptr);

	//Fill D
	fptr = fopen("./example/D.txt","r");
	for(i=0;i<nDis*pHorizon;i++)
	{
		fscanf(fptr,"%f", &num);
		D[i] = num;
	}
	fclose(fptr);

	//Fill x
	fptr = fopen("./example/x.txt","r");
	for(i=0;i<nState;i++)
	{
		fscanf(fptr,"%f", &num);
		x[i] = num;
	}
	fclose(fptr);
}

int main()
{
	// QP is of parametric from 
	// J(U) = min U 1/2*U'QpU + Fp'U + 1/2*Mp
	// st GpU <= Kp
	
	hipDeviceReset();

	int N, M;

	M = pHorizon*nInput;
	N = 4*pHorizon*nInput;

	// host matrix
	float *hQp_inv = newMatrix(M,M);
	float *hQp = newMatrix(M,M);

	float *hFp1;
	float *hFp2;
	float *hFp3;

	float *hMp1;
	float *hMp2;
	float *hMp3;
	float *hMp4;
	float *hMp5;
	float *hMp6;

	float *hFp = newMatrix(nInput*pHorizon,1);
	float *hMp = newMatrix(1,1);
	float *hGp;
	float *hKp;
	float *hx;
	float *hD; 
	float *htheta; 
	float *hZ; 

	hFp1 = newMatrix(nInput*pHorizon, nDis*pHorizon);
	hFp2 = newMatrix(nInput*pHorizon, nState);
	hFp3 = newMatrix(1, nInput*pHorizon);
	hMp1 = newMatrix(nState, nState);
	hMp2 = newMatrix(nDis*pHorizon, nState);
	hMp3 = newMatrix(nDis*pHorizon, nDis*pHorizon);
	hMp4 = newMatrix(1, nState);
	hMp5 = newMatrix(1, nDis*pHorizon);
	hMp6 = newMatrix(1,1);
	hGp = newMatrix(4*pHorizon*nInput, nInput*pHorizon);
	hKp = newMatrix(1,4*pHorizon*nInput);
	hZ = newMatrix(nOutput*pHorizon, nState);
	htheta = newMatrix(nOutput*pHorizon, nDis*pHorizon);
	hD = newMatrix(nDis*pHorizon,1);
	hx = newMatrix(nState, 1);

	// device matrix
	float *Qp_inv = newMatrixCUDA(M,M);
	float *Qp = newMatrixCUDA(M,M);

	float *Fp1;
	float *Fp2;
	float *Fp3;

	float *Mp1;
	float *Mp2;
	float *Mp3;
	float *Mp4;
	float *Mp5;
	float *Mp6;

	float *Fp = newMatrixCUDA(nInput*pHorizon,1);
	float *Mp = newMatrixCUDA(1,1);
	float *Gp;
	float *Kp;
	float *x;
	float *D; 
	float *theta; 
	float *Z; 

	Fp1 = newMatrixCUDA(nInput*pHorizon, nDis*pHorizon);
	Fp2 = newMatrixCUDA(nInput*pHorizon, nState);
	Fp3 = newMatrixCUDA(1, nInput*pHorizon);
	Mp1 = newMatrixCUDA(nState, nState);
	Mp2 = newMatrixCUDA(nDis*pHorizon, nState);
	Mp3 = newMatrixCUDA(nDis*pHorizon, nDis*pHorizon);
	Mp4 = newMatrixCUDA(1, nState);
	Mp5 = newMatrixCUDA(1, nDis*pHorizon);
	Mp6 = newMatrixCUDA(1,1);
	Gp = newMatrixCUDA(4*pHorizon*nInput, nInput*pHorizon);
	Kp = newMatrixCUDA(1,4*pHorizon*nInput);
	Z = newMatrixCUDA(nOutput*pHorizon, nState);
	theta = newMatrixCUDA(nOutput*pHorizon, nDis*pHorizon);
	D = newMatrixCUDA(nDis*pHorizon,1);
	x = newMatrixCUDA(nState, 1);	

	input(hQp_inv, hFp1, hFp2, hFp3, hMp1, hMp2, hMp3, hMp4, hMp5, hMp6, hGp, hKp, hx, hD, htheta, hZ);
	Gauss_Jordan(hQp_inv, hQp, M);
	copyToDevice(Qp_inv, hQp_inv, M, M);
	copyToDevice(Qp, hQp, M, M);
	copyToDevice(Fp1, hFp1, nInput*pHorizon, nDis*pHorizon);
	copyToDevice(Fp2, hFp2, nInput*pHorizon, nState);
	copyToDevice(Fp3, hFp3, 1, nInput*pHorizon);
	copyToDevice(Mp1, hMp1, nState, nState);
	copyToDevice(Mp2, hMp2, nDis*pHorizon, nState);
	copyToDevice(Mp3, hMp3, nDis*pHorizon, nDis*pHorizon);
	copyToDevice(Mp4, hMp4, 1, nState);
	copyToDevice(Mp5, hMp5, 1, nDis*pHorizon);
	copyToDevice(Mp6, hMp6, 1,1);
	copyToDevice(Gp, hGp, 4*pHorizon*nInput, nInput*pHorizon);
	copyToDevice(Kp, hKp, 1,4*pHorizon*nInput);
	copyToDevice(Z, hZ, nOutput*pHorizon, nState);
	copyToDevice(D, hD, nDis*pHorizon,1);
	copyToDevice(theta, htheta, nOutput*pHorizon, nDis*pHorizon);
	copyToDevice(x, hx, nState, 1);

	computeFp(Fp, Fp1, Fp2, Fp3, D, x);
	computeMp(Mp, Mp1, Mp2, Mp3, Mp4, Mp5, Mp6, D, x);

	// matrices and vectors required for dual form of QP
	float *Qd = newMatrixCUDA(N,N);
	float *Fd = newMatrixCUDA(N,1);
	float *Md = newMatrixCUDA(1,1);	
	float *Y  = newMatrixCUDA(N,1);
	float *U  = newMatrixCUDA(M,1);

	convertToDual(Qd, Fd, Md, Qp_inv, Gp, Kp, Fp, Mp, N, M);

	solveQuadraticDual(Y, Qd, Fd, Md, U, Qp, Qp_inv, Fp, Mp, Gp, Kp, N, M);

	computeUfromY(U, Y, Fp, Gp, Qp_inv, N, M);

	float Jp = computeCost(U, Qp, Fp, Mp, M);
	float Jd = computeCost(Y, Qd, Fd, Md, N);

	printf("Jp = %f\n", Jp);
	printf("Jd = %f\n", Jd);
	
	float *hU = newMatrix(M,1);
	float *hY = newMatrix(N,1);

	copyToHost(hU,U,M,1);
	copyToHost(hY,Y,N,1);

	printf("Printing U*\n");
	for(int i=0;i<M;i++)
	{
		printf("\t%f\n", hU[i]);
	}

	free(hQp_inv);
	free(hQp);
	free(hFp1);
	free(hFp2);
	free(hFp3);
	free(hMp1);
	free(hMp2);
	free(hMp3);
	free(hMp4);
	free(hMp5);
	free(hMp6); 
	free(hFp);
	free(hMp);
	free(hGp);
	free(hKp);
	free(hx);
	free(hD);
	free(htheta);
	free(hZ);

	hipFree(Qp_inv);
	hipFree(Qp);
	hipFree(Fp1);
	hipFree(Fp2);
	hipFree(Fp3);
	hipFree(Mp1);
	hipFree(Mp2);
	hipFree(Mp3);
	hipFree(Mp4);
	hipFree(Mp5);
	hipFree(Mp6); 
	hipFree(Fp);
	hipFree(Mp);
	hipFree(Gp);
	hipFree(Kp);
	hipFree(x);
	hipFree(D);
	hipFree(theta);
	hipFree(Z);
	
	hipFree(Qd);
	hipFree(Fd);
	hipFree(Md);
	hipFree(Y);
	hipFree(U);
}
